#include "hip/hip_runtime.h"
#include <glm/glm.hpp>
#include <hiprand/hiprand_kernel.h>
#include "methods.h"

namespace Kernels
{
    constexpr int BLOCK_SAMPLE_COUNT{5};
    constexpr int PIXEL_SAMPLE_COUNT{1};
    constexpr int CLOSEST_COUNT{4};

    namespace Constants
    {
        __constant__ int intConstants[IntConstantIDs::INT_CONSTANTS_COUNT];
        __device__ int2 imgRes(){return {intConstants[IntConstantIDs::IMG_RES_X], intConstants[IntConstantIDs::IMG_RES_Y]};}
        __device__ int2 colsRows(){return{intConstants[IntConstantIDs::COLS], intConstants[IntConstantIDs::ROWS]};}
        __device__ int gridSize(){return intConstants[IntConstantIDs::GRID_SIZE];} 
        __constant__ void* dataPointers[DataPointersIDs::POINTERS_COUNT];
        __device__ hipSurfaceObject_t* surfaces(){return reinterpret_cast<hipSurfaceObject_t*>(dataPointers[DataPointersIDs::SURFACES]);}
        __device__ hipTextureObject_t* textures(){return reinterpret_cast<hipTextureObject_t*>(dataPointers[DataPointersIDs::TEXTURES]);} 
        
        __constant__ float floatConstants[FloatConstantIDs::FLOAT_CONSTANTS_COUNT];
        __device__ float2 scanRange(){return {floatConstants[FloatConstantIDs::SCAN_RANGE_START], floatConstants[FloatConstantIDs::SCAN_RANGE_END]};}
        __device__ float scanRangeSize(){return floatConstants[FloatConstantIDs::SCAN_RANGE_SIZE];}
        __device__ float2 pixelSize(){return {floatConstants[FloatConstantIDs::PX_SIZE_X], floatConstants[FloatConstantIDs::PX_SIZE_Y]};}
        __device__ float2 halfPixelSize(){return {floatConstants[FloatConstantIDs::PX_SIZE_X_HALF], floatConstants[FloatConstantIDs::PX_SIZE_Y_HALF]};}

        __device__ constexpr int MAX_IMAGES{4};
        __constant__ float weights[MAX_IMAGES];
        __constant__ float2 offsets[MAX_IMAGES];
        __constant__ int2 blockOffsets[BLOCK_OFFSET_COUNT];
    }

    //extern __shared__ half localMemory[];

    __device__ float4 operator+ (const float4  &a, const float4 &b)
    {
        float4 result;
        result.x = a.x + b.x;
        result.y = a.y + b.y;
        result.z = a.z + b.z;
        return result;
    }
    
    __device__ float4 operator/ (const float4  &a, const float &b)
    {
        float4 result;
        result.x = a.x/b;
        result.y = a.y/b;
        result.z = a.z/b;
        return result;
    }
    
    __device__ float4 operator* (const float  &a, const float4 &b)
    {
        float4 result;
        result.x = a*b.x;
        result.y = a*b.y;
        result.z = a*b.z;
        return result;
    }

    __device__ bool coordsOutside(int2 coords, int2 resolution)
    {
        return (coords.x >= resolution.x || coords.y >= resolution.y);
    }

    __device__ int2 getImgCoords()
    {
        int2 coords;
        constexpr int SUBSAMPLING{2};
        coords.x = (threadIdx.x + blockIdx.x * blockDim.x)*SUBSAMPLING;
        coords.y = (threadIdx.y + blockIdx.y * blockDim.y)*SUBSAMPLING;
        return coords;
    }

    __device__ float2 normalizeCoords(int2 coords)
    {
        auto res = Constants::imgRes();
        return {static_cast<float>(coords.x)/res.x,
                static_cast<float>(coords.y)/res.y};
    }
   
    namespace Pixel
    {
        __device__ float distance(float3 &a, float3 &b)
        {
            float dist = fmaxf(fmaxf(fabsf(a.x-b.x), fabsf(a.y-b.y)), fabsf(a.z-b.z));
            return dist;
        }

        __device__ void store(float4 px, int imageID, int2 coords)
        {
            surf2Dwrite<float4>(px, Constants::surfaces()[imageID], coords.x*sizeof(float4), coords.y);
        } 
        
        __device__ float4 load(int imageID, int2 coords)
        {
            int id = Constants::textures()[imageID];
            float2 halfPx = Constants::halfPixelSize(); 
            return tex2D<float4>(id, coords.x+halfPx.x, coords.y+halfPx.y);
        }
    }
 
        class ElementRange
        {
            private:
            float3 minCol{FLT_MAX, FLT_MAX, FLT_MAX};
            float3 maxCol{FLT_MIN, FLT_MIN, FLT_MIN};
            
            public:
            __device__ void add(float4 val)
            {
                minCol.x = fminf(minCol.x, val.x);
                minCol.y = fminf(minCol.y, val.y);
                minCol.z = fminf(minCol.z, val.z);
                maxCol.x = fmaxf(maxCol.x, val.x);
                maxCol.y = fmaxf(maxCol.y, val.y);
                maxCol.z = fmaxf(maxCol.z, val.z);
            }
            __device__ float dispersionAmount()
            {
                return Pixel::distance(minCol, maxCol); 
            }      
            __device__ ElementRange& operator+=(const float4& rhs){

              add(rhs);
              return *this;
            }
        };

    __device__ int2 focusCoords(int gridID, int2 pxCoords, float focus)
    {
        float2 offset = Constants::offsets[gridID];
        //return {static_cast<int>(round(offset.x*focus+pxCoords.x)), static_cast<int>(round(offset.y*focus+pxCoords.y))};
        return {__float2int_rn(__fmaf_rn(offset.x, focus, pxCoords.x)),__float2int_rn(__fmaf_rn(offset.y, focus, pxCoords.y))};
    }

    namespace FocusLevel
    {      
        __device__ void evaluateBlock(int gridID, float focus, int2 coords, ElementRange *dispersions)
        {
            for(int blockPx=0; blockPx<BLOCK_OFFSET_COUNT; blockPx++)
            {
                int2 offset = Constants::blockOffsets[blockPx]; 
                int2 inBlockCoords{coords.x+offset.x, coords.y+offset.y};
                auto px{Pixel::load(gridID, focusCoords(gridID, inBlockCoords, focus))};
                dispersions[blockPx] += px;
            }
        }

        __device__ float evaluateDispersion(int2 coords, float focus)
        {
            auto cr = Constants::colsRows();
            ElementRange dispersionCalc[BLOCK_OFFSET_COUNT];
                
            int gridID = 0;
            for(int row=0; row<cr.y; row++) 
            {     
                gridID = row*cr.x;
                for(int col=0; col<cr.x; col++) 
                {
                    evaluateBlock(gridID, focus, coords, dispersionCalc);
                    gridID++;
                }
            } 
            float finalDispersion{0};
            for(int blockPx=0; blockPx<BLOCK_OFFSET_COUNT; blockPx++)
                finalDispersion += dispersionCalc[blockPx].dispersionAmount();
            return finalDispersion;
        }
 
        __device__ float4 render(int2 coords, float focus)
        {
            auto cr = Constants::colsRows();
            float4 sum;
            int gridID = 0; 
          
                auto weights = Constants::weights;
                for(int row=0; row<cr.y; row++) 
                {     
                    gridID = row*cr.x;
                    for(int col=0; col<cr.x; col++) 
                    {
                        auto px{Pixel::load(gridID, focusCoords(gridID, coords, focus))};
                        sum = sum + weights[gridID] * px;
                        gridID++;
                    }
                }
            return sum;
        }      
    }
    
    namespace Focusing
    {    
        class Optimum
        {
            public:
            float optimalFocus{0};
            float minDispersion{FLT_MAX};
            __device__ bool add(float focus, float dispersion)
            {
                if(dispersion < minDispersion)
                {
                   minDispersion = dispersion;
                   optimalFocus = focus; 
                   return true;
                }
                return false;
            }
            __device__ void addForce(float focus, float dispersion)
            {
                   minDispersion = dispersion;
                   optimalFocus = focus; 
            }
        }; 

        __device__ Optimum& minOpt(Optimum &a, Optimum &b)
        {
            if(a.minDispersion < b.minDispersion)
                return a;
            else
                return b;
        }

        __device__ float bruteForce(int2 coords)
        {
            int steps = 32;
            float stepSize{static_cast<float>(Constants::scanRangeSize())/steps};
            float focus{Constants::scanRange().x};
            Optimum optimum;
            
            for(int step=0; step<steps; step++)
            {
                float dispersion = FocusLevel::evaluateDispersion(coords, focus);
                optimum.add(focus, dispersion);
                focus += stepSize;  
            }
            return optimum.optimalFocus;
        }

    }

    __global__ void process()
    {
        int2 coords = getImgCoords();
        if(coordsOutside(coords, Constants::imgRes()))
            return;
 
        for(int pixelID = 0; pixelID < 4; pixelID++)
        {
            int2 pixelCoords{coords.x+pixelID/2, coords.y+pixelID%2};
            float4 color = {0,0,0,0};
            for(int row=0; row<Constants::colsRows().y; row++) 
            {     
                int i = row*Constants::colsRows().x;
                for(int col=0; col<Constants::colsRows().x; col++) 
                {
                    color = color + Pixel::load(i, focusCoords(i, pixelCoords, Constants::scanRange().x));
                    i++;
                }
            }
            color = color / 4.0f;
            color.w = 1.0;
            Pixel::store(color, FileNames::RENDER_IMAGE, pixelCoords);
        }
    }
}
